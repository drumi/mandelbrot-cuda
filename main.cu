#include "hip/hip_runtime.h"
#include "bmp.h"

#include <iostream>
#include <string>
#include <complex>
#include <thread>
#include <vector>
#include <atomic>

struct Complex
{
    float real;
    float imag;
};

namespace 
{
    namespace DEFAULT
    {
        namespace FLAG
        {
            char const* const FILE_NAME     = "-o";
            char const* const IMAGE_WIDTH   = "-w";
            char const* const IMAGE_HEIGHT  = "-h";
            char const* const GRANULARITY   = "-g";
            char const* const THREADS_COUNT = "-t";
            char const* const ZOOM_LEVEL    = "-z";
            char const* const POINT_ORIGIN  = "-p";
            char const* const ITERATIONS    = "-c";
        }   

        namespace IMAGE
        {
            char const* NAME = "mandelbrot.bmp";
            int const WIDTH = 3840;
            int const HEIGHT = 2160;
            float const ZOOM_LEVEL = 1.0;
            int const BYTES_PER_PIXEL = 3;
            Complex const POINT_ORIGIN = {0, 0};

            uint8_t const TINT_ON_ESCAPE = 32;
        }

        namespace THREADS
        {
            int const GRANULARITY = 1;
            int const ITERATIONS = 256;
            int const COUNT = std::thread::hardware_concurrency();
        }

        float const INFINITY_THRESHOLD = 4.0;
    }
}

#include <chrono>
namespace
{
    class Clock
    {
        std::chrono::time_point<std::chrono::high_resolution_clock> start = std::chrono::high_resolution_clock::now();

        public:
        int getElapsedMilliseconds() const
        {
            std::chrono::time_point<std::chrono::high_resolution_clock> now = std::chrono::high_resolution_clock::now();
            return std::chrono::duration_cast<std::chrono::milliseconds>(now - start).count();
        }
    };
}


struct ProgramParameters
{
    int    imageWidth      = DEFAULT::IMAGE::WIDTH;
    int    imageHeight     = DEFAULT::IMAGE::HEIGHT;
    int    threadsCount    = DEFAULT::THREADS::COUNT;
    int    granularity     = DEFAULT::THREADS::GRANULARITY;
    int    iterationsCount = DEFAULT::THREADS::ITERATIONS;
    float zoomLevel       = DEFAULT::IMAGE::ZOOM_LEVEL;

    char const* imageOutputName = DEFAULT::IMAGE::NAME;
    Complex pointOrigin = DEFAULT::IMAGE::POINT_ORIGIN;
};

struct ThreadParameters
{
    int chunkSize;
    int chunksCount;
    int remainderChunkSize;
    int imageTotalSize;
    int bytesPerWidth;
    int bytesPerHeight;

    float dx;
    float dy;

    Complex bottomLeftCoordinates;
    Complex upperRightCoordinates;
};

void printExecutingParameters(ProgramParameters const p)
{
    std::cout << '\n'
              << DEFAULT::FLAG::FILE_NAME     << " for file name in bmp format.  " << "Executing: " << DEFAULT::FLAG::FILE_NAME     <<" " << p.imageOutputName    << "\n"
              << DEFAULT::FLAG::IMAGE_WIDTH   << " for image width.              " << "Executing: " << DEFAULT::FLAG::IMAGE_WIDTH   <<" " << p.imageWidth         << "\n"
              << DEFAULT::FLAG::IMAGE_HEIGHT  << " for image height.             " << "Executing: " << DEFAULT::FLAG::IMAGE_HEIGHT  <<" " << p.imageHeight        << "\n"
              << DEFAULT::FLAG::GRANULARITY   << " for granularity.              " << "Executing: " << DEFAULT::FLAG::GRANULARITY   <<" " << p.granularity        << "\n"
              << DEFAULT::FLAG::THREADS_COUNT << " for thread count.             " << "Executing: " << DEFAULT::FLAG::THREADS_COUNT <<" " << p.threadsCount       << "\n"
              << DEFAULT::FLAG::ZOOM_LEVEL    << " for image zoom.               " << "Executing: " << DEFAULT::FLAG::ZOOM_LEVEL    <<" " << p.zoomLevel          << "\n"
              << DEFAULT::FLAG::ITERATIONS    << " for complex iterations count. " << "Executing: " << DEFAULT::FLAG::ITERATIONS    <<" " << p.iterationsCount    << "\n"
              << DEFAULT::FLAG::POINT_ORIGIN  << " for image center point.       " << "Executing: " << DEFAULT::FLAG::POINT_ORIGIN  <<" " << p.pointOrigin.real << " " << p.pointOrigin.imag << "\n"
              << '\n';
}

ProgramParameters handleInput(int argc, const char** argv)
{
    ProgramParameters result;

    int i = 1;
    while(i < argc)
    {
        std::string inputFlag(argv[i]);

        if(inputFlag == DEFAULT::FLAG::FILE_NAME)
        {
            //result.imageOutputName = argv[i+1].c_str();
            i += 2;
        }
        else if(inputFlag == DEFAULT::FLAG::IMAGE_WIDTH)
        {
            result.imageWidth = atoi(argv[i+1]);
            i += 2;
        }
        else if(inputFlag == DEFAULT::FLAG::IMAGE_HEIGHT)
        {
            result.imageHeight = atoi(argv[i+1]);
            i += 2;
        }
        else if(inputFlag == DEFAULT::FLAG::GRANULARITY)
        {
            result.granularity = atoi(argv[i+1]);
            i += 2;
        }
        else if(inputFlag == DEFAULT::FLAG::THREADS_COUNT) 
        {
           // result.threadsCount = atoi(argv[i+1]);
            i += 2;;
        }
        else if(inputFlag == DEFAULT::FLAG::ZOOM_LEVEL) 
        {
            result.zoomLevel = atof(argv[i+1]);
            i += 2;
        }
        else if(inputFlag == DEFAULT::FLAG::POINT_ORIGIN)
        {
            result.pointOrigin = {(float)atof(argv[i+1]), (float)atof(argv[i+2])};
            i += 3;
        }
        else if(inputFlag == DEFAULT::FLAG::ITERATIONS)
        {
            result.iterationsCount = atoi(argv[i+1]);
            i += 2;
        }
        else
        {
            std::cerr << "Invalid parameter supplied: " << argv[i] << '\n';
            exit(-1);
        }
    }

    return result;
}

ThreadParameters generateThreadParameters(ProgramParameters const p)
{
    ThreadParameters result;

    int const totalPixels = p.imageHeight * p.imageWidth;

    result.imageTotalSize     = totalPixels * DEFAULT::IMAGE::BYTES_PER_PIXEL; 
    result.chunkSize          = (totalPixels / (p.granularity * p.threadsCount)) * DEFAULT::IMAGE::BYTES_PER_PIXEL;
    result.chunksCount        = result.imageTotalSize / result.chunkSize;
    result.remainderChunkSize = result.imageTotalSize % result.chunkSize;
    
    float const zoom = 2.0 / p.zoomLevel;
    float const aspectRatio = p.imageHeight / (float) p.imageWidth;

    result.bottomLeftCoordinates = {-zoom + p.pointOrigin.real, -zoom * aspectRatio + p.pointOrigin.imag};
    result.upperRightCoordinates = { zoom + p.pointOrigin.real,  zoom * aspectRatio + p.pointOrigin.imag};
    
    result.bytesPerWidth = p.imageWidth * DEFAULT::IMAGE::BYTES_PER_PIXEL;
    result.bytesPerHeight = p.imageHeight;

    result.dx = (result.upperRightCoordinates.real - result.bottomLeftCoordinates.real);
    result.dy = (result.upperRightCoordinates.imag - result.bottomLeftCoordinates.imag);

    return result;
}

__device__
int computeSteps(int const iterations, float real, float imag)
{
    float currR = real;
    float currI = imag;

    float squaredR = currR * currR;
    float squaredI = currI * currI;

    for (int i = 1; i <= iterations; ++i)
    {
        float const growthIndex = squaredR + squaredR;

        if(growthIndex > 4.0)
            return i;

        currI = 2 * currI * currR + imag;
        currR = squaredR - squaredI + real;

        squaredR = currR * currR;
        squaredI = currI * currI;
    }

    return 0;
}

__device__
void computePortionOfImage(int const imageStartIndex, int const imageEndIndex, int const iterations, ThreadParameters const t, uint8_t* rawImage)
{
    for(int i = imageStartIndex; i < imageEndIndex; i += 3)
    {
        int const y = i / t.bytesPerWidth;
        int const x = i % t.bytesPerWidth;

        float const realFraction = (x / (float)t.bytesPerWidth);
        float const imagFraction = (y / (float)t.bytesPerHeight);

        float const real = realFraction * t.dx + t.bottomLeftCoordinates.real;
        float const imag = imagFraction * t.dy + t.bottomLeftCoordinates.imag;

        int const steps = computeSteps(iterations, real, imag);
        uint8_t const color = (255 * steps) /iterations;

        rawImage[i  ] = 32 * (steps != 0); // b
        rawImage[i+1] = color; // g
        rawImage[i+2] = 0;     // r; 0 by default
    }
}

__global__
void computeImage(ProgramParameters const p, ThreadParameters const t, uint8_t* rawImage)
{
    int const threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int currentChunkNumber = threadId - p.threadsCount;

    // Handle normal chunks
    while ((currentChunkNumber += p.threadsCount) < t.chunksCount)
    {
        int const imageStartIndex = currentChunkNumber * t.chunkSize;
        int const imageEndIndex = (currentChunkNumber + 1) * t.chunkSize - 1;

        computePortionOfImage(imageStartIndex, imageEndIndex, p.iterationsCount, t, rawImage);
    }

    // Handle remainder
    if(t.remainderChunkSize != 0 && (currentChunkNumber == t.chunksCount))
    {
        int const imageStartIndex = currentChunkNumber * t.chunkSize;
        int const imageEndIndex = t.imageTotalSize - 1;

        computePortionOfImage(imageStartIndex, imageEndIndex, p.iterationsCount, t, rawImage);
    }
}

int main(int const argc, const char** argv) 
{
    Clock const programClock;

    ProgramParameters programParameters = handleInput(argc, argv);
    programParameters.threadsCount = 16384;
    
    printExecutingParameters(programParameters);
    
    ThreadParameters const threadParameters = generateThreadParameters(programParameters);
    ProgramParameters p = programParameters;

    uint8_t* rawImage = nullptr;
    uint8_t* rawImageHost = nullptr;

    rawImageHost = new uint8_t[p.imageWidth * p.imageHeight * DEFAULT::IMAGE::BYTES_PER_PIXEL]();
    hipMalloc(&rawImage, p.imageWidth * p.imageHeight * DEFAULT::IMAGE::BYTES_PER_PIXEL);

    computeImage<<<128,128>>>(programParameters, threadParameters, rawImage);

    hipMemcpy(rawImageHost, rawImage, p.imageWidth * p.imageHeight * DEFAULT::IMAGE::BYTES_PER_PIXEL, hipMemcpyDeviceToHost);

    BMPImage::save(programParameters.imageOutputName, programParameters.imageHeight, programParameters.imageWidth, rawImageHost);
    
    hipFree(rawImage);
    delete[] rawImageHost;

    std::cout << "Total time for program execution: " << programClock.getElapsedMilliseconds() << "ms\n";

    return 0;
}
